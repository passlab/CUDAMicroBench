#include "hip/hip_runtime.h"
//******************************************************************************************************************//
// Copyright (c) 2021, University of North Carolina at Charlotte
// and Lawrence Livermore National Security, LLC.
// SPDX-License-Identifier: (BSD-3-Clause)
//*****************************************************************************************************************//
#include "warpDivergenceTest.h"


__global__ void warmingup(float *x, float *y, float *z) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid % 2 == 0) {
        z[tid] = 2 * x[tid] + 3 * y[tid];

    } else {
        z[tid] = 3 * x[tid] + 2 * y[tid];
    }
}

__global__ void warpDivergence(float *x, float *y, float *z) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a = 2, b = 3;
    if (tid % 2 != 0) {
        a = 3;
        b = 2;
    }
    z[tid] = a * x[tid] + b * y[tid];
}

__global__ void noWarpDivergence(float *x, float *y, float *z) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int even = tid % 2 == 0;
    float a = even * 2 + (1 - even) * 3;
    float b = even * 3 + (1 - even) * 2;
    z[tid] = a * x[tid] + b * y[tid];
}
void warpDivergenceTest_cuda(REAL* x, REAL* y, REAL *warp_divergence, REAL *no_warp_divergence, int n) {
  REAL *d_x, *d_y, *d_warp_divergence, *d_no_warp_divergence;
  hipMalloc(&d_x, n*sizeof(REAL));
  hipMalloc(&d_y, n*sizeof(REAL));
  hipMalloc(&d_warp_divergence, n*sizeof(REAL));
  hipMalloc(&d_no_warp_divergence, n*sizeof(REAL));

  hipMemcpy(d_x, x, n*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n*sizeof(REAL), hipMemcpyHostToDevice);

  hipDeviceSynchronize();

  warmingup<<<(n+255)/256, 256>>> (d_x, d_y, d_warp_divergence);
  hipDeviceSynchronize();

  warpDivergence<<<(n+255)/256, 256>>>(d_x, d_y, d_warp_divergence);
  hipDeviceSynchronize();

  noWarpDivergence<<<(n+255)/256, 256>>>(d_x, d_y, d_no_warp_divergence);
  hipDeviceSynchronize();

  hipMemcpy(warp_divergence, d_warp_divergence, n*sizeof(REAL), hipMemcpyDeviceToHost);
  hipMemcpy(no_warp_divergence, d_no_warp_divergence, n*sizeof(REAL), hipMemcpyDeviceToHost);


  hipFree(d_x);
  hipFree(d_y);

  hipFree(d_warp_divergence);
  hipFree(d_no_warp_divergence);


}
